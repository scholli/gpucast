#include "hip/hip_runtime.h"
/********************************************************************************
*
* Copyright (C) 2009-2011 Bauhaus University Weimar
*
*********************************************************************************
*
*  module     : octree_raycasting.cu
*  project    : gpucast
*  description:
*
********************************************************************************/

// CUDA includes 
#include <gpucast/volume/cuda_map_resources.hpp> 
 
#include <cmath> 
#include <iostream>
#include <gpucast/gl/util/timer.hpp>

#include <cuda_types.h> 
#include <cuda_globals.h>
#include <device_types.h>

#include <raycast_octree.h>
#include <cuda_renderconfig.h>
#include <get_kernel_workitems.h>

namespace octree
{
  surface<void, cudaSurfaceType2D> out_color_image;
  surface<void, cudaSurfaceType2D> out_depth_image;
  surface<void, cudaSurfaceType2D> in_external_image;
  surface<void, cudaSurfaceType2D> in_position_image;
}

///////////////////////////////////////////////////////////////////////////////
extern "C" __kernel void raycast_octree_kernel ( gpucast::renderconfig  config,
                                                 gpucast::bufferinfo    info,
                                                 float4 const*          matrixbuffer,
                                                 uint4 const*           nodebuffer,
                                                 uint4 const*           facelistbuffer,
                                                 float4 const*          bboxbuffer,
                                                 float const*           limitbuffer,
                                                 uint4 const*           surfacedatabuffer,
                                                 float4 const*          surfacepointsbuffer,
                                                 float4 const*          volumedatabuffer,
                                                 float4 const*          volumepointsbuffer,
                                                 float4 const*          attributedatabuffer,
                                                 float2 const*          attributepointsbuffer )
{
  int sx = blockIdx.x*blockDim.x + threadIdx.x;
	int sy = blockIdx.y*blockDim.y + threadIdx.y;

  if ( sx >= config.width || sy >= config.height )
	{ 
		return;
	} 

  int2  coords      = int2_t(sx, sy);

  raycast_octree( config,
                  info,
                  coords,
                  matrixbuffer,
                  nodebuffer,
                  facelistbuffer,
                  bboxbuffer,
                  limitbuffer,
                  surfacedatabuffer,
                  surfacepointsbuffer,
                  volumedatabuffer,
                  volumepointsbuffer,
                  attributedatabuffer,
                  attributepointsbuffer,
                  octree::out_color_image,
                  octree::out_depth_image,
                  octree::in_position_image,
                  octree::in_external_image );
}


///////////////////////////////////////////////////////////////////////////////
extern "C" void invoke_octree_raycasting_kernel ( gpucast::renderconfig const& config,
                                                  gpucast::bufferinfo const&   info,
                                                  struct hipGraphicsResource* input_position_resource,
                                                  struct hipGraphicsResource* colorbuffer_resource,
                                                  struct hipGraphicsResource* depthbuffer_resource, 
                                                  struct hipGraphicsResource* external_color_depth_resource,
                                                  struct hipGraphicsResource* cuda_octree_node_buffer,
                                                  struct hipGraphicsResource* cuda_octree_face_buffer,
                                                  struct hipGraphicsResource* cuda_octree_bbox_buffer,
                                                  struct hipGraphicsResource* cuda_octree_limit_buffer,
                                                  struct hipGraphicsResource* cuda_surface_data_buffer,
                                                  struct hipGraphicsResource* cuda_surface_points_buffer,
                                                  struct hipGraphicsResource* cuda_volume_data_buffer,
                                                  struct hipGraphicsResource* cuda_volume_points_buffer,
                                                  struct hipGraphicsResource* cuda_attribute_data_buffer,
                                                  struct hipGraphicsResource* cuda_attribute_points_buffer,
                                                  struct hipGraphicsResource* cuda_matrixbuffer )
{ 
  hipGraphicsResource* cuda_resources[] = {
                                             input_position_resource,
                                             colorbuffer_resource,
                                             depthbuffer_resource,
                                             external_color_depth_resource,
                                             cuda_octree_node_buffer,
                                             cuda_octree_face_buffer,
                                             cuda_octree_bbox_buffer,
                                             cuda_octree_limit_buffer,
                                             cuda_surface_data_buffer,
                                             cuda_surface_points_buffer,
                                             cuda_volume_data_buffer,
                                             cuda_volume_points_buffer,
                                             cuda_attribute_data_buffer,
                                             cuda_attribute_points_buffer,
                                             cuda_matrixbuffer
                                           };
  
  map_resources ( sizeof ( cuda_resources ) / sizeof ( hipGraphicsResource* ), cuda_resources );
   
  // map output image
  bind_mapped_resource_to_surface ( colorbuffer_resource,           &octree::out_color_image );
  bind_mapped_resource_to_surface ( depthbuffer_resource,           &octree::out_depth_image );
  bind_mapped_resource_to_surface ( input_position_resource,        &octree::in_position_image );
  bind_mapped_resource_to_surface ( external_color_depth_resource,  &octree::in_external_image );

  uint4*  nodebuffer;
  uint4*  facebuffer;
  float4* bboxbuffer;
  float*  limitbuffer;
  uint4*  surfacedatabuffer;
  float4* surfacepointsbuffer;
  float4* volumedatabuffer;
  float4* volumepointsbuffer;
  float4* attributedatabuffer;
  float2* attributepointsbuffer;
  float4* matrixbuffer;

  bind_mapped_resource_to_pointer ( cuda_octree_node_buffer,      nodebuffer            );
  bind_mapped_resource_to_pointer ( cuda_octree_face_buffer,      facebuffer            );
  bind_mapped_resource_to_pointer ( cuda_octree_bbox_buffer,      bboxbuffer            );
  bind_mapped_resource_to_pointer ( cuda_octree_limit_buffer,     limitbuffer           );
  bind_mapped_resource_to_pointer ( cuda_surface_data_buffer,     surfacedatabuffer     );
  bind_mapped_resource_to_pointer ( cuda_surface_points_buffer,   surfacepointsbuffer   );
  bind_mapped_resource_to_pointer ( cuda_volume_data_buffer,      volumedatabuffer      );
  bind_mapped_resource_to_pointer ( cuda_volume_points_buffer,    volumepointsbuffer    );
  bind_mapped_resource_to_pointer ( cuda_attribute_data_buffer,   attributedatabuffer   );
  bind_mapped_resource_to_pointer ( cuda_attribute_points_buffer, attributepointsbuffer );
  bind_mapped_resource_to_pointer ( cuda_matrixbuffer,            matrixbuffer          );

  { // raycast kernel
    int workitems     = int(get_kernel_workitems ( &raycast_octree_kernel ));

    int workgroups_x  = config.width  + (workitems - config.width % workitems); 
    int workgroups_y  = config.height + (workitems - config.height % workitems);

    dim3 block  ( workitems, workitems, 1 );
    dim3 grid   ( workgroups_x / block.x, workgroups_y / block.y, 1);

    // execute kernel  
    raycast_octree_kernel<<< grid, block>>> ( config,
                                              info,
                                              matrixbuffer, 
                                              nodebuffer, 
                                              facebuffer, 
                                              bboxbuffer, 
                                              limitbuffer,
                                              surfacedatabuffer,
                                              surfacepointsbuffer,
                                              volumedatabuffer,
                                              volumepointsbuffer,
                                              attributedatabuffer,
                                              attributepointsbuffer);
  }

  // unmap gl-resources
  unmap_resources( sizeof ( cuda_resources ) / sizeof ( hipGraphicsResource* ), cuda_resources );
}
