#include "hip/hip_runtime.h"
/********************************************************************************
*
* Copyright (C) 2009-2013 Bauhaus University Weimar
*
*********************************************************************************
*
*  module     : sample_fragmentlists.cu
*  project    : gpucast
*  description:
*
********************************************************************************/
#include <cmath> 
#include <iostream>

#include <gpucast/volume/cuda_map_resources.hpp> 

// C cuda includes
#include <cuda_types.h> 
#include <device_types.h>

#include <get_kernel_workitems.h>
#include <sample_fragmentlists.h>

// project includes

#include <gpucast/gl/util/timer.hpp>
  /*
///////////////////////////////////////////////////////////////////////////////
extern "C" __kernel void sort_kernel    ( int         width, 
                                          int         height,
                                          int2        tilesize,
                                          unsigned    pagesize,
                                          uint4*      indexlist )
{
  int sx = blockIdx.x*blockDim.x + threadIdx.x;
	int sy = blockIdx.y*blockDim.y + threadIdx.y;

  if ( sx >= width || sy >= height )
	{ 
		return;
	} 

  int2  coords        = int2_t(sx, sy);

  int fragindex       = 0; 
  unsigned nfragments = 0;

  surf2Dread ( &fragindex,  fraglist::in_headpointer_image, coords.x*sizeof(float), coords.y );
  surf2Dread ( &nfragments, fraglist::in_fragmentcount_image, coords.x*sizeof(float), coords.y );

  //uint nfragments = count_fragments(indexlist, fragindex);
   
  bubble_sort_indexlist_1 ( fragindex, nfragments, indexlist ); 
}
 */

namespace unified_sampling
{
  surface<void, cudaSurfaceType2D> out_color_image;
  surface<void, cudaSurfaceType2D> out_depth_image;
  surface<void, cudaSurfaceType2D> in_headpointer_image;
  surface<void, cudaSurfaceType2D> in_fragmentcount_image;
  surface<void, cudaSurfaceType2D> in_external_image;
}

///////////////////////////////////////////////////////////////////////////////
extern "C" __kernel void sample_fragmentlist_kernel (  int            width,  
                                                       int            height,
                                                       int2           tilesize,
                                                       unsigned       pagesize,
                                                       unsigned       volume_info_offset,
                                                       float          nearplane,
                                                       float          farplane,
                                                       float3         background, 
                                                       float          iso_threshold,
                                                       int            show_isosides,
                                                       int            adaptive_sampling,
                                                       float          min_sample_distance,
                                                       float          max_sample_distance,
                                                       float          adaptive_sample_scale,
                                                       int            screenspace_newton_error,
                                                       float          fixed_newton_epsilon, 
                                                       unsigned       max_iterations_newton,
                                                       unsigned       max_steps_binary_search,
                                                       float          global_attribute_min,
                                                       float          global_attribute_max,  
                                                       float          surface_transparency, 
                                                       float          isosurface_transparency, 
                                                       float4 const*  matrices, 
                                                       uint4*         indexlist,
                                                       uint4 const*   surfacedatabuffer, 
                                                       float4 const*  surfacepointsbuffer, 
                                                       float4 const*  volumedatabuffer, 
                                                       float4 const*  volumepointsbuffer, 
                                                       float4 const*  attributedatabuffer,
                                                       float2 const*  attributepointsbuffer
                                          )
{
  int sx = blockIdx.x*blockDim.x + threadIdx.x;
	int sy = blockIdx.y*blockDim.y + threadIdx.y;

  if ( sx >= width || sy >= height )
	{ 
		return;
	}

  int2  coords      = int2_t(sx, sy);

  clock_t t0 = clock();  
    
  sample_fragmentlists ( threadIdx.x,
                         width, 
                         height, 
                         coords,  
                         nearplane,
                         farplane,
                         background,
                         tilesize,
                         pagesize,
                         volume_info_offset,
                         iso_threshold,
                         show_isosides,
                         adaptive_sampling,
                         min_sample_distance,
                         max_sample_distance, 
                         adaptive_sample_scale,
                         screenspace_newton_error,
                         fixed_newton_epsilon, 
                         max_iterations_newton,
                         max_steps_binary_search,
                         global_attribute_min, 
                         global_attribute_max, 
                         surface_transparency, 
                         isosurface_transparency,
                         matrices, 
                         indexlist,  
                         surfacedatabuffer,
                         surfacepointsbuffer,
                         volumedatabuffer, 
                         volumepointsbuffer, 
                         attributedatabuffer, 
                         attributepointsbuffer, 
                         unified_sampling::out_color_image,
                         unified_sampling::out_depth_image,
                         unified_sampling::in_headpointer_image,  
                         unified_sampling::in_external_image );
     
  clock_t t1 = clock();  
   
  float relative_costs = float_t(t1 - t0) / 10000000.0;   
       
  float4 costs_color = transferfunction ( relative_costs );  
  //surf2Dwrite ( costs_color, out_color_image, coords.x*sizeof(float4), coords.y );
}   


///////////////////////////////////////////////////////////////////////////////
extern "C" void invoke_unified_sampling  ( unsigned                     width,
                                           unsigned                     height,
                                           int2                         tilesize, 
                                           unsigned                     pagesize,
                                           unsigned                     volume_info_offset,
                                           float                        nearplane,
                                           float                        farplane,
                                           float3                       background, 
                                           float                        iso_threshold,
                                           int                          show_isosides,
                                           int                          adaptive_sampling,
                                           float                        min_sample_distance, 
                                           float                        max_sample_distance,
                                           float                        adaptive_sample_scale,
                                           int                          screenspace_newton_error,  
                                           float                        fixed_newton_epsilon,  
                                           unsigned                     max_iterations_newton,
                                           unsigned                     max_steps_binary_search,
                                           float                        global_attribute_min,
                                           float                        global_attribute_max,
                                           float                        surface_transparency,
                                           float                        isosurface_transparency,  
                                           struct hipGraphicsResource* matrices_resource,
                                           struct hipGraphicsResource* colorbuffer_resource,
                                           struct hipGraphicsResource* depthbuffer_resource, 
                                           struct hipGraphicsResource* headpointer_resource, 
                                           struct hipGraphicsResource* fragmentcount_resource,  
                                           struct hipGraphicsResource* indexlist_resource, 
                                           struct hipGraphicsResource* surface_data_buffer_resource,
                                           struct hipGraphicsResource* surface_points_buffer_resource,
                                           struct hipGraphicsResource* volume_data_buffer_resource,
                                           struct hipGraphicsResource* volume_points_buffer_resource,
                                           struct hipGraphicsResource* attribute_data_buffer_resource,
                                           struct hipGraphicsResource* attribute_points_buffer_resource,
                                           struct hipGraphicsResource* external_color_depth_resource )
{ 
  hipGraphicsResource* cuda_resources[] = { matrices_resource     ,
                                             colorbuffer_resource  ,
                                             depthbuffer_resource  ,
                                             headpointer_resource  ,
                                             fragmentcount_resource,
                                             indexlist_resource    ,
                                             surface_data_buffer_resource,
                                             surface_points_buffer_resource,
                                             volume_data_buffer_resource, 
                                             volume_points_buffer_resource,
                                             attribute_data_buffer_resource,
                                             attribute_points_buffer_resource,
                                             external_color_depth_resource
                                           };

  map_resources ( sizeof ( cuda_resources ) / sizeof ( hipGraphicsResource* ), cuda_resources );
   
  // map output image
  bind_mapped_resource_to_surface ( colorbuffer_resource,           &unified_sampling::out_color_image );
  bind_mapped_resource_to_surface ( depthbuffer_resource,           &unified_sampling::out_depth_image );
  bind_mapped_resource_to_surface ( headpointer_resource,           &unified_sampling::in_headpointer_image );
  bind_mapped_resource_to_surface ( fragmentcount_resource,         &unified_sampling::in_fragmentcount_image );
  bind_mapped_resource_to_surface ( external_color_depth_resource,  &unified_sampling::in_external_image );

  // retrieve device pointer for mapped buffers  
  uint4*      indexlist;
  float4*     matrices; 
  uint4*      surfacedatabuffer; 
  float4*     surfacepointsbuffer;
  float4*     volumedatabuffer; 
  float4*     volumepointsbuffer; 
  float4*     attributedatabuffer; 
  float2*     attributepointsbuffer;

  bind_mapped_resource_to_pointer ( matrices_resource        , matrices        );
  bind_mapped_resource_to_pointer ( indexlist_resource       , indexlist       );

  bind_mapped_resource_to_pointer ( surface_data_buffer_resource     , surfacedatabuffer     );
  bind_mapped_resource_to_pointer ( surface_points_buffer_resource   , surfacepointsbuffer   );
  bind_mapped_resource_to_pointer ( volume_data_buffer_resource      , volumedatabuffer      );
  bind_mapped_resource_to_pointer ( volume_points_buffer_resource    , volumepointsbuffer    );
  bind_mapped_resource_to_pointer ( attribute_data_buffer_resource   , attributedatabuffer   );
  bind_mapped_resource_to_pointer ( attribute_points_buffer_resource , attributepointsbuffer );

  float memsettime;

  hipEvent_t start, stop; 
   
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // single sort pass
#if 0 
  hipEventRecord(start,0); 
  hipDeviceSynchronize();

  { // sort kernel
    std::size_t workitems     = get_kernel_workitems ( "sort_kernel" );

    std::size_t workgroups_x  = width  + (workitems - width % workitems); 
    std::size_t workgroups_y  = height + (workitems - height % workitems);

    dim3 block  ( workitems, workitems, 1 );
    dim3 grid   ( workgroups_x / block.x, workgroups_y / block.y, 1);

    sort_kernel<<< grid, block>>> ( width,  
                                    height, 
                                    tilesize,
                                    pagesize, 
                                    indexlist );
  } 

  hipEventRecord(stop,0); 
  hipDeviceSynchronize();

  hipEventElapsedTime(&memsettime, start, stop); 
  std::cout << " sort kernel ms : " << memsettime; 
#endif

  hipEventRecord(start,0); 

  { // raycast kernel
    std::size_t workitems     = get_kernel_workitems ( &sample_fragmentlist_kernel );
    //std::size_t workitems     = 8;

    //std::cout << workitems << " workitems per block.\n";

    std::size_t workgroups_x  = width  + (workitems - width % workitems); 
    std::size_t workgroups_y  = height + (workitems - height % workitems);

    dim3 block  ( workitems, workitems, 1 );
    dim3 grid   ( workgroups_x / block.x, workgroups_y / block.y, 1);

     
    // execute kernel  
    sample_fragmentlist_kernel<<< grid, block>>> ( width,  
                                                   height, 
                                                   tilesize,
                                                   pagesize,
                                                   volume_info_offset, 
                                                   nearplane,  
                                                   farplane,
                                                   background, 
                                                   iso_threshold,
                                                   show_isosides,
                                                   adaptive_sampling, 
                                                   min_sample_distance, 
                                                   max_sample_distance,  
                                                   adaptive_sample_scale,
                                                   screenspace_newton_error,
                                                   fixed_newton_epsilon,
                                                   max_iterations_newton,
                                                   max_steps_binary_search,
                                                   global_attribute_min,
                                                   global_attribute_max,
                                                   surface_transparency,
                                                   isosurface_transparency,   
                                                   matrices, 
                                                   indexlist,
                                                   surfacedatabuffer, 
                                                   surfacepointsbuffer, 
                                                   volumedatabuffer, 
                                                   volumepointsbuffer,
                                                   attributedatabuffer,
                                                   attributepointsbuffer);

  } 

  hipEventRecord(stop, 0);   
  hipDeviceSynchronize();

  hipEventElapsedTime(&memsettime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // unmap gl-resources
  unmap_resources( sizeof ( cuda_resources ) / sizeof ( hipGraphicsResource* ), cuda_resources );
}



 